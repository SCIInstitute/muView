#include "hip/hip_runtime.h"
/***************************************************************************
                                  cuda_knn.h
                             -------------------
                               W. Michael Brown

  K-nearest neighbors routines for CUDA

 __________________________________________________________________________
    This file is part of the Math Library
 __________________________________________________________________________

    begin                : Wed Jan 28 2009
    copyright            : (C) 2009 by W. Michael Brown
    email                : wmbrown@sandia.gov
 ***************************************************************************/

#include <iostream>
#include "cuda_calls.h"
#include "cuda_knn.h"
#include "cuda_timer.h"
#include "cuda_device.h"
#include "cuda_knn_texture.h"

using namespace std;


template void cuda_kneighbors(const float *in, const cu_sizet k, 
                              cu_sizet *neighbors, float *dists, 
              							  const cu_sizet rows, const cu_sizet cols);

#ifdef CUDA_DOUBLE
template void cuda_kneighbors(const double *in, const cu_sizet k, 
                              cu_sizet *neighbors, double *dists, 
							                const cu_sizet rows, const cu_sizet cols);
#endif

// Numer of threads for 1D blocks
#define BLOCK_1D 64
// Sqrt(number of threads) for 2D blocks 
#define BLOCK_2D 16

// 1D block adjustment for float and double precision
template <class numbr> inline int cuda_knn_block1d() { return 1; }
template <> inline int cuda_knn_block1d<double>() { return 4; }


/***************************************************************************
 * Compute the symmetric distance matrix for the input data
 * 
 * dist - N x N matrix of distances to be computed (diagonal is infinity)
 * d - Dimensionality of the input data
 * dist_pitch - Size of a row in the matrix in memory
 ***************************************************************************/
template<class numbr>
__global__ void kernel_knn(numbr *dist, const int d, const size_t dist_pitch) {
  // ti and tj index the threads within the block
  int ti=threadIdx.x;
  int tj=threadIdx.y;
  // vdim is the number of datapoint dimensions that can fit in shared memory
  int vdim=blockDim.y;
  
  // i and j index the two datapoints being compared
  int i=ti+blockIdx.x*blockDim.x;
  int j=tj+blockIdx.y*vdim;
  
  // Each block stores datapoints indexed by all threads in the block
  __shared__ numbr in_blocki[BLOCK_2D][BLOCK_2D+1];
  __shared__ numbr in_blockj[BLOCK_2D][BLOCK_2D+1];
  
  // If i==j set to inf, else, to 0
  numbr distance;
  if (i==j)
    distance=cuda_inf<numbr>();
  else
    distance=cuda_zero<numbr>();
	
  for (int l=0; l<d; l+=vdim) {
    // Each thread loads two elements of shared input matrices
    in_blocki[ti][tj]=cuda_knn_tex2D<numbr>(i,l+tj);
    in_blockj[tj][ti]=cuda_knn_tex2D<numbr>(j,l+ti);
	// Loop over the number of dimensions that fit in shared memory
	int loop_end=vdim;
	if (l+loop_end>d)
	  loop_end=d-l;
    __syncthreads();
	// Compute distance between i and j
    for (int k=0; k<loop_end; ++k) {
      numbr d2=in_blocki[ti][k]-in_blockj[tj][k];
	  distance+=d2*d2;
	}
	// Sync so that we don't load new 'vdim' set of dimensions too soon
    __syncthreads();
  }

  // Store distance in global memory
  dist[INT_MUL(j,dist_pitch)+i]=distance;
}

/***************************************************************************
 * Sort to find k closest distances and corresponding datapoint indices
 * O(n*k)
 * 
 * sort - k+1 x N matrix containing k-closest distances in sorted order
 * nbor - k+1 x N matrix containing indices of k-closest neighbors
 * dist - N x N matrix of distances between datapoints
 * k - number of neighbors
 * n - number of datapoints (N)
 * sort_pitch, dist_pitch, nbor_pitch - row sizes of each supplied matrix
 ***************************************************************************/
template<class numbr>
__global__ void kernel_knnsort(numbr *sort, int *nbor, const numbr *dist, 
                               const int k, const int n, const size_t sort_pitch,
							   const size_t dist_pitch, const size_t nbor_pitch){
  // i indexes the datapoint whose neighbors are being sorted
  int i=threadIdx.x+blockIdx.x*blockDim.x;

  // Minimum distance of Inf to start with
  int loop_end=INT_MUL(k+1,sort_pitch);
  for (int z=i; z<loop_end; z+=sort_pitch)
	sort[z]=cuda_inf<numbr>();
  
  // Insertion sort O(n*k)
  const numbr *d_row=dist+i;
  for (int z=0; z<n; ++z) {
    numbr *c=sort+INT_MUL(k-1,sort_pitch)+i;
	int *d=nbor+INT_MUL(k-1,nbor_pitch)+i;
    for (int y=0; y<k && *d_row<*c; ++y) {
	  *(c+sort_pitch)=*c;
  	  *(d+nbor_pitch)=*d;
	  c-=sort_pitch;
	  d-=nbor_pitch;
	}
	*(c+sort_pitch)=*d_row;
	*(d+nbor_pitch)=z;
	d_row+=dist_pitch;
  }
  
  for (int z=i; z<loop_end; z+=sort_pitch)
    sort[z]=sqrt(sort[z]);
}

template<class numbr>
void cuda_kneighbors(const numbr *in, const cu_sizet k, cu_sizet *neighbors,
                     numbr *dists, const cu_sizet rows, const cu_sizet cols) {
  cout << endl << "Calculating k-neighbors on GPU: " << get_cuda_device_name() 
       << endl;
  
  // Timing stuff
  #ifdef CUDA_TIME
  TimerCUDA timer;
  timer.start();
  #endif
  
  // Determine the block size and grid size
  const int BX=BLOCK_2D;
  const int BY=BLOCK_2D;
  // Adjust 1D grid size if we are using double precision
  const int B1D=BLOCK_1D/cuda_knn_block1d<numbr>();
  int GX=static_cast<int>(ceil(static_cast<double>(cols)/
                               static_cast<double>(BX)));
  int GY=static_cast<int>(ceil(static_cast<double>(cols)/
                               static_cast<double>(BY)));
  int GX2=static_cast<int>(ceil(static_cast<double>(cols)/
                                static_cast<double>(B1D)));

  // Calculate matrix dimensions that give even division of work
  // - Matrices are allocated with extra room for load-balancing
  cu_sizet v_width=max(GX*BX,GX2*B1D);
  cu_sizet v_height=max(GY*BY,GX2*B1D);
  cu_sizet v_dims=rows;
  if (rows%BY!=0)
    v_dims+=(BY-rows%BY);

  // Allocate memory on device for input matrix and distance matrix
  numbr *dev_dist;
  hipArray* dev_in;
  size_t dist_pitch;

  // Create read-only texture for input matrix and copy data to device
  hipChannelFormatDesc channel;
  cuda_knn_get_channel<numbr>(channel);
  CUDA_SAFE_CALL(hipMallocArray(&dev_in, &channel, v_width, v_dims));
  for (int i=0; i<rows; i++)
    CUDA_SAFE_CALL(hipMemcpyToArray(dev_in, 0, i, in+i*cols, cols*sizeof(numbr),
	                                 hipMemcpyHostToDevice));
  knn_bind_texture<numbr>(dev_in,&channel);

  // Reserve memory for pairwise distance matrix on device
  CUDA_SAFE_CALL(hipMallocPitch((void **)&dev_dist,&dist_pitch,
                                 v_width*sizeof(numbr),v_height));

  #ifdef CUDA_TIME
  timer.next();
  printf("malloc and memcpy:\t%f\n", timer.time());
  #endif

  // Calculate pairwise distance matrix
  dim3 dimBlock(BX,BY);
  dim3 dimGrid(GX,GY);
  kernel_knn<numbr><<<dimGrid,dimBlock>>>(dev_dist,rows,
                                          dist_pitch/sizeof(numbr));
  #ifdef CUDA_TIME
  timer.next();
  printf("knn:\t\t\t%f\n", timer.time());
  #endif

  // Free memory from input matrix and reserve memory for sorted distances
  // and neighbor indices
  knn_unbind_texture<numbr>();
  hipFreeArray(dev_in);
  numbr *dev_sort;
  size_t sort_pitch, nbor_pitch;
  int *dev_nbor;
  CUDA_SAFE_CALL(hipMallocPitch((void **)&dev_nbor,&nbor_pitch,
                                 v_width*sizeof(int),k+1));
  CUDA_SAFE_CALL(hipMallocPitch((void **)&dev_sort,&sort_pitch,
                                 v_width*sizeof(numbr),k+1));

  #ifdef CUDA_TIME
  timer.next();
  printf("malloc:\t\t\t%f\n", timer.time());
  #endif

  kernel_knnsort<numbr><<<GX2,B1D>>>(dev_sort,dev_nbor,dev_dist,k,cols,
                                     sort_pitch/sizeof(numbr),
									 dist_pitch/sizeof(numbr),
									 nbor_pitch/sizeof(int));
  #ifdef CUDA_TIME
  timer.next();
  printf("sort:\t\t\t%f\n", timer.time());
  #endif

  // Copy distances and neighbors back to host
  CUDA_SAFE_CALL(hipMemcpy2D(dists,cols*sizeof(numbr),dev_sort,sort_pitch,
                              cols*sizeof(numbr),k,hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy2D(neighbors,cols*sizeof(int),dev_nbor,nbor_pitch,
                              cols*sizeof(int),k,hipMemcpyDeviceToHost));

  // Free allocated memory
  hipFree(dev_nbor);
  hipFree(dev_dist);
  #ifdef CUDA_TIME
  timer.stop();
  printf("memcpy and free:\t%f\n", timer.time());
  #endif
}
