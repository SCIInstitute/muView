/***************************************************************************
                               cuda_device.cu
                             -------------------
                               W. Michael Brown

  Utilities for dealing with cuda devices

 __________________________________________________________________________
    This file is part of the Math Library
 __________________________________________________________________________

    begin                : Wed Jan 28 2009
    copyright            : (C) 2009 by W. Michael Brown
    email                : wmbrown@sandia.gov
 ***************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "cuda_calls.h"
#include "cuda_device.h"

bool choose_cuda_device(int &num) {
  int deviceCount;
  CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));
  if (deviceCount == 0)
	return false;
  
  num = 0;
  int major = 0, minor = 0, mp = 0;
  
  hipDeviceProp_t deviceProp;
  for (int i=0; i<deviceCount; i++) {
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, i));
    if (i == 0 && deviceProp.major == 9999 && deviceProp.minor == 9999) {
      printf("There is no device supporting CUDA\n");
	  return false;
	} 
	
	bool better=false;
	if (deviceProp.major>=major) {
	  if (deviceProp.major>major)
	    better=true;
	  else if (deviceProp.minor>=minor) {
	    if (deviceProp.minor>minor)
		  better=true;
		else if (deviceProp.multiProcessorCount>mp)
		  better=true;
      }
    }
	
	if (better) {
	  num=i;
	  major=deviceProp.major;
	  minor=deviceProp.minor;
	  mp=deviceProp.multiProcessorCount;
	}
  }
  return true;
}

/// Set the CUDA device to the specified device number
void set_cuda_device(int num) {
  CUDA_SAFE_CALL(hipSetDevice(num));
}

/// Get the current CUDA device name
string get_cuda_device_name() {
  int num;
  CUDA_SAFE_CALL(hipGetDevice(&num));
  string name;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, num));
  name=deviceProp.name;
  return name;
}
