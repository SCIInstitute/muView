#include "hip/hip_runtime.h"

#include <heart/cuda_common.h>
#include <Data/Mesh/TetMesh.h>
#include <SCI/Vex3.h>

#define BLOCK_SIZE 256

/*
union Point {
    struct {
        float x, y, z;
    };
    float data[3];
};


__device__ inline Point Difference(Point p0, Point p1){
    Point ret;
    ret.x = p0.x-p1.x;
    ret.y = p0.y-p1.y;
    ret.z = p0.z-p1.z;
    return ret;
}

__device__ inline float Distance(Point p0, Point p1){
    Point tmp = Difference( p0, p1 );
    return sqrt( tmp.x*tmp.x + tmp.y*tmp.y + tmp.z*tmp.z );
}


*/


__host__ __device__ inline SCI_Vex3 SCI_cross(const SCI_Vex3& v1, const SCI_Vex3& v2){
    SCI_Vex3 ret;
    ret.x = v1.y*v2.z - v1.z*v2.y;
    ret.y = v1.z*v2.x - v1.x*v2.z;
    ret.z = v1.x*v2.y - v1.y*v2.x;
    return ret;
}

__host__ __device__ inline SCI_Vex3 SCI_subtract(const SCI_Vex3& v1, const SCI_Vex3& v2){
    SCI_Vex3 ret;
    ret.x = v1.x - v2.x;
    ret.y = v1.y - v2.y;
    ret.z = v1.z - v2.z;
    return ret;
}

__host__ __device__ inline float SCI_dot(const SCI_Vex3& v1, const SCI_Vex3& v2){
        return ((v1.x*v2.x) + (v1.y*v2.y) + (v1.z*v2.z));
}



//__device__ bool isPointInsideTet( SCI_Vex3 t0, SCI_Vex3 t1, SCI_Vex3 t2, SCI_Vex3 t3, SCI_Vex3 p ){
__device__ bool isPointInsideTet( SCI_Vex3 * v, SCI_Vex3 p ){
    for(int i = 0; i < 4; i++){
        SCI_Vex3 a = v[i];
        SCI_Vex3 ba = SCI_subtract( v[(i+1)%4], a );
        SCI_Vex3 ca = SCI_subtract( v[(i+2)%4], a );
        SCI_Vex3 da = SCI_subtract( v[(i+3)%4], a );
        SCI_Vex3 n = SCI_cross( ba, ca );
        if( SCI_dot(n, da ) * SCI_dot(n, SCI_subtract(p,a) ) < 0 ) return false;
    }
    return true;
}

__global__ void voxel_associativity_kernel(Data_Mesh_Tetmesh * tets, int tetN, SCI_Vex3 * points, int pointN, SCI_Vex3 * vox_centers, int * vox_assoc, int vox_dim){

    SCI_Vex3 tmpPoints[4];

    int thrd = threadIdx.x;

    int vox_id = blockIdx.x * 256 + threadIdx.x;

    for(int i = 0; i < tetN; i++ ){

        // load (up to) the next 32 tets into shared memory
        tmpPoints[0] = points[ tets[ i ].data[0] ];
        tmpPoints[1] = points[ tets[ i ].data[1] ];
        tmpPoints[2] = points[ tets[ i ].data[2] ];
        tmpPoints[3] = points[ tets[ i ].data[3] ];

        if( isPointInsideTet( tmpPoints, vox_centers[vox_id] ) ){
            vox_assoc[vox_id] = i;
        }
    }

    /*
    for(int i = 0; i < tetN; i++ ){

        // load (up to) the next 32 tets into shared memory
        if( (i+thrd/4) < tetN ){
            tmpPoints[thrd] = points[ tets[ i+thrd/4 ].data[thrd%4] ];
        }
        __syncthreads( );

        for( int j = 0; j < 128/4 && i < tetN; j++, i++ ){
            //if( isPointInsideTet( tmpPoints[4*j+0], tmpPoints[4*j+1], tmpPoints[4*j+2], tmpPoints[4*j+3], vox_centers[vox_id] ) ){
            if( isPointInsideTet( tmpPoints, 4*j, vox_centers[vox_id] ) ){
                vox_assoc[vox_id] = i;
            }
        }
    }
    */

    //vox_assoc[vox_id] = 0;
}

__global__ void kernel(int *array)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  array[index] = 7;
}

extern "C"
void voxel_associativity( Data_Mesh_Tetmesh * h_tets, int tetN, SCI_Vex3 * h_points, int pointN, SCI_Vex3 * h_vox_centers, int * h_vox_assoc, int vox_dim ){

    Data_Mesh_Tetmesh * d_tets;
    SCI_Vex3          * d_points;
    int               * d_vox_assoc;
    SCI_Vex3          * d_vox_centers;

    int dev = findCudaDevice( 0, 0 );
    if( dev == -1 ) {
        return;
    }

    printf("%i",vox_dim);

    checkCudaErrors( hipMalloc( (void**) &d_tets,        sizeof( Data_Mesh_Tetmesh ) * tetN                        ) );
    checkCudaErrors( hipMalloc( (void**) &d_points,      sizeof( SCI_Vex3 )          * pointN                      ) );
    checkCudaErrors( hipMalloc( (void**) &d_vox_assoc,   sizeof( int )               * vox_dim * vox_dim * vox_dim ) );
    checkCudaErrors( hipMalloc( (void**) &d_vox_centers, sizeof( SCI_Vex3 )          * vox_dim * vox_dim * vox_dim ) );

    checkCudaErrors( hipMemcpy( d_tets,        h_tets,        sizeof( Data_Mesh_Tetmesh ) * tetN,                        hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_points,      h_points,      sizeof( SCI_Vex3 )          * pointN,                      hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_vox_assoc,   h_vox_assoc,   sizeof( int )               * vox_dim * vox_dim * vox_dim, hipMemcpyHostToDevice ) );
    checkCudaErrors( hipMemcpy( d_vox_centers, h_vox_centers, sizeof( SCI_Vex3 )          * vox_dim * vox_dim * vox_dim, hipMemcpyHostToDevice ) );

    int block_size = 256;
    int grid_size  = vox_dim*vox_dim*vox_dim/256;
    printf("Launching kernel\n"); fflush(stdout);
    //voxel_associativity_kernel<<< grid_size, block_size >>>( d_tets, tetN, d_points, pointN, d_vox_centers, d_vox_assoc, vox_dim );
    for(int tet = 0; tet < 128; tet+=128){
        voxel_associativity_kernel<<< grid_size, block_size >>>( d_tets+tet, 128, d_points, pointN, d_vox_centers, d_vox_assoc, vox_dim );
    }

    checkCudaErrors( hipMemcpy( h_vox_assoc, d_vox_assoc, sizeof( int ) * vox_dim * vox_dim * vox_dim,  hipMemcpyDeviceToHost ) );

    checkCudaErrors( hipFree( d_tets        ) );
    checkCudaErrors( hipFree( d_points      ) );
    checkCudaErrors( hipFree( d_vox_assoc   ) );
    checkCudaErrors( hipFree( d_vox_centers ) );

    hipDeviceReset();


    /*
    int num_elements = 256;

    int num_bytes = num_elements * sizeof(int);

    // pointers to host & device arrays
    int *device_array = 0;
    int *host_array = 0;

    // malloc a host array
    host_array = (int*)malloc(num_bytes);

    // hipMalloc a device array
    hipMalloc((void**)&device_array, num_bytes);

    int block_size = 128;
    int grid_size = num_elements / block_size;

    kernel<<<grid_size,block_size>>>(device_array);

    // download and inspect the result on the host:
    hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

    // print out the result element by element
    for(int i=0; i < num_elements; ++i)
    {
      printf("%d ", host_array[i]);
    }

    // deallocate memory
    free(host_array);
    hipFree(device_array);
    */

}
